#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <algorithm> //sort
#include <vector>
#include <iostream>
#include "MedianFilter.h"

__device__ void Sort(unsigned char* array, int size)
{
	int i, key, j;
	for (i = 1; i < size; i++)
	{
		key = array[i];
		j = i - 1;
		while (j >= 0 && array[j] > key)
		{
			array[j + 1] = array[j];
			j = j - 1;
		}
		array[j + 1] = key;
	}
}


__global__ void MedianFilterSharedMemoryKernel(unsigned char* InputImage, unsigned char* OutputImage, int width, int height)
{
	__shared__ unsigned char ImagePixels[18][18];
	unsigned char PixelVals[9];
	int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	int y = (blockIdx.y*blockDim.y) + threadIdx.y;
	int threadID = (y*width)+x;
	if(x < width && y < height)
	{
		
		ImagePixels[threadIdx.y+1][threadIdx.x+1] = InputImage[threadID];
		//elements of left and right columns,top and bottom rows of grid
		if(x == 0)
		{
			ImagePixels[threadIdx.y+1][threadIdx.x] = 0;
		}
		if(x == width-1)
		{
			ImagePixels[threadIdx.y+1][threadIdx.x+2] = 0;
		}
		if(y == 0)
		{
			ImagePixels[threadIdx.y][threadIdx.x+1] = 0;
		}
		if(y == height-1)
		{
			ImagePixels[threadIdx.y+2][threadIdx.x+1] = 0;
		}
		//corner elements of grid
		if(x == 0 && y == 0)
		{
			ImagePixels[threadIdx.y][threadIdx.x] = 0;
		}
		if(x == 0 && y == height-1)
		{
			ImagePixels[threadIdx.y+2][threadIdx.x] = 0;
		}
		if(x == width-1 && y == 0)
		{
			ImagePixels[threadIdx.y][threadIdx.x+2] = 0;
		}
		if(x == width-1 && y == height-1)
		{
			ImagePixels[threadIdx.y+2][threadIdx.x+2] = 0;
		}

		//left and right columns
		if(threadIdx.x == 0 && x != 0)
		{
			ImagePixels[threadIdx.y+1][threadIdx.x] = InputImage[threadID-1];
		}
		if(threadIdx.x == 15 && x != width-1)
		{
			ImagePixels[threadIdx.y+1][threadIdx.x+2] = InputImage[threadID+1];
		}

		//top and bottom rows
		if(threadIdx.y == 0 && y != 0)
		{
			ImagePixels[threadIdx.y][threadIdx.x+1] = InputImage[threadID-width];
		}
		if(threadIdx.y == 15 && y != height-1)
		{
			ImagePixels[threadIdx.y+2][threadIdx.x+1] = InputImage[threadID+width];
		}

		//corner elements
		if(threadIdx.x == 0 && threadIdx.y == 0 && x != 0 && y != 0)
		{
			ImagePixels[threadIdx.y][threadIdx.x] = InputImage[threadID-width-1];
		}
		if(threadIdx.x == 0 && threadIdx.y == 15 && x != 0 && y != height-1)
		{
			ImagePixels[threadIdx.y+2][threadIdx.x] = InputImage[threadID+width-1];
		}
		if(threadIdx.x == 15 && threadIdx.y == 0 && x != width-1 && y != 0)
		{
			ImagePixels[threadIdx.y][threadIdx.x+2] = InputImage[threadID-width+1];
		}
		if(threadIdx.x == 15 && threadIdx.y == 15 && x != width-1 && y != height-1)
		{
			ImagePixels[threadIdx.y+2][threadIdx.x+2] = InputImage[threadID+width+1];
		}
		__syncthreads();
		
		PixelVals[0] = ImagePixels[threadIdx.y+1][threadIdx.x+1];
		PixelVals[1] = ImagePixels[threadIdx.y+1][threadIdx.x];
		PixelVals[2] = ImagePixels[threadIdx.y+1][threadIdx.x+2];
		PixelVals[3] = ImagePixels[threadIdx.y][threadIdx.x+1];
		PixelVals[4] = ImagePixels[threadIdx.y][threadIdx.x];
		PixelVals[5] = ImagePixels[threadIdx.y][threadIdx.x+2];
		PixelVals[6] = ImagePixels[threadIdx.y+2][threadIdx.x+1];
		PixelVals[7] = ImagePixels[threadIdx.y+2][threadIdx.x];
		PixelVals[8] = ImagePixels[threadIdx.y+2][threadIdx.x+2];
		if(x == 0 || y == 0 || x == width-1 || y == height-1)
		{
			if(y>0 && x>0){PixelVals[4] = InputImage[threadID-width-1];}
			else{PixelVals[4] = 0;}
			if(y>0 && x<(width-1)){PixelVals[5] = InputImage[threadID-width+1];}
			else{PixelVals[5] = 0;}
			if(y<(height-1) && x>0){PixelVals[7] = InputImage[threadID+width-1];}
			else{PixelVals[7] = 0;}
			if(y<(height-1) && x<(width-1)){PixelVals[8] = InputImage[threadID+width+1];}
			else{PixelVals[8] = 0;}
		}
		Sort(PixelVals,9);
		OutputImage[threadID] = PixelVals[4];
	}
}

__global__ void MedianFilterKernel(unsigned char* InputImage, unsigned char* OutputImage, int width, int height)
{
	unsigned char PixelVals[9];
	int x = (blockIdx.x*blockDim.x) + threadIdx.x;;
	int y = (blockIdx.y*blockDim.y) + threadIdx.y;
	int threadID = (y*width)+x;
	if(x < width && y < height)
	{
		PixelVals[0] = InputImage[threadID];
		if(x>0)
		{
			PixelVals[1] = InputImage[threadID-1];
		}
		else
		{
			PixelVals[1] = 0;
		}
		if(x<(width-1))
		{
			PixelVals[2] = InputImage[threadID+1];
		}
		else
		{
			PixelVals[2] = 0;
		}
		if(y>0)
		{
			PixelVals[3] = InputImage[threadID-width];
		}
		else
		{
			PixelVals[3] = 0;
		}
		if(y>0 && x>0)
		{
			PixelVals[4] = InputImage[threadID-width-1];
		}
		else
		{
			PixelVals[4] = 0;
		}
		if(y>0 && x<(width-1))
		{
			PixelVals[5] = InputImage[threadID-width+1];
		}
		else
		{
			PixelVals[5] = 0;
		}
		if(y<(height-1))
		{
			PixelVals[6] = InputImage[threadID+width];
		}
		else
		{
			PixelVals[6] = 0;
		}
		if(y<(height-1) && x>0)
		{
			PixelVals[7] = InputImage[threadID+width-1];
		}
		else
		{
			PixelVals[7] = 0;
		}
		if(y<(height-1) && x<(width-1))
		{
			PixelVals[8] = InputImage[threadID+width+1];
		}
		else
		{
			PixelVals[8] = 0;
		}
		Sort(PixelVals,9);
		OutputImage[threadID] = PixelVals[4];
	}

}
// C Function to run matrix multiplication kernel
bool MedianFilterGPU( Bitmap* image, Bitmap* outputImage, bool sharedMemoryUse )
{

	hipError_t status;
	int width = image->Width();
	int height = image->Height();

	int bytes = width * height * sizeof(char);
	int index =0;

	unsigned char *Md, *Pd;
	hipMalloc((void**) &Md, bytes);
	hipMalloc((void**) &Pd, bytes);

	hipMemcpy(Md, image->image, bytes, hipMemcpyHostToDevice);

	dim3 dimBlock(16, 16); 
	dim3 dimGrid((int)ceil((float)width/16), (int)ceil((float)height/16));
	if(sharedMemoryUse == false)
	{
		MedianFilterSharedMemoryKernel<<<dimGrid, dimBlock>>>(Md, Pd, width, height);
	}
	else
	{
		MedianFilterKernel<<<dimGrid, dimBlock>>>(Md, Pd, width, height);
	}
	// Wait for completion
	hipDeviceSynchronize();
	// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) 
	{
		std::cout << "Kernel failed: " << hipGetErrorString(status) <<
		std::endl;
		hipFree(Md);
		hipFree(Pd);
		return false;
	}
	// Retrieve the result matrix
	hipMemcpy(outputImage->image, Pd, bytes, hipMemcpyDeviceToHost);
	index = 0;
	int pcount = 0;
	index = 0;
	hipFree(Md);
	hipFree(Pd);
	// Success
	return true;
}
